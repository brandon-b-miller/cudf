#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/detail/utilities/device_operators.cuh>
#include <cudf/strings/udf/case.cuh>
#include <cudf/strings/udf/char_types.cuh>
#include <cudf/strings/udf/replace.cuh>
#include <cudf/strings/udf/search.cuh>
#include <cudf/strings/udf/starts_with.cuh>
#include <cudf/strings/udf/strip.cuh>
#include <cudf/strings/udf/udf_string.cuh>

#include <cuda/atomic>

#include <hip/hip_cooperative_groups.h>

#include <limits>
#include <type_traits>

using namespace cudf::strings::udf;

extern "C" __device__ int len(int* nb_retval, void const* str)
{
  auto sv    = reinterpret_cast<cudf::string_view const*>(str);
  *nb_retval = sv->length();
  return 0;
}

extern "C" __device__ int startswith(bool* nb_retval, void const* str, void const* substr)
{
  auto str_view    = reinterpret_cast<cudf::string_view const*>(str);
  auto substr_view = reinterpret_cast<cudf::string_view const*>(substr);

  *nb_retval = starts_with(*str_view, *substr_view);
  return 0;
}

extern "C" __device__ int endswith(bool* nb_retval, void const* str, void const* substr)
{
  auto str_view    = reinterpret_cast<cudf::string_view const*>(str);
  auto substr_view = reinterpret_cast<cudf::string_view const*>(substr);

  *nb_retval = ends_with(*str_view, *substr_view);
  return 0;
}

extern "C" __device__ int contains(bool* nb_retval, void const* str, void const* substr)
{
  auto str_view    = reinterpret_cast<cudf::string_view const*>(str);
  auto substr_view = reinterpret_cast<cudf::string_view const*>(substr);

  *nb_retval = (str_view->find(*substr_view) != cudf::string_view::npos);
  return 0;
}

extern "C" __device__ int find(int* nb_retval, void const* str, void const* substr)
{
  auto str_view    = reinterpret_cast<cudf::string_view const*>(str);
  auto substr_view = reinterpret_cast<cudf::string_view const*>(substr);

  *nb_retval = str_view->find(*substr_view);
  return 0;
}

extern "C" __device__ int rfind(int* nb_retval, void const* str, void const* substr)
{
  auto str_view    = reinterpret_cast<cudf::string_view const*>(str);
  auto substr_view = reinterpret_cast<cudf::string_view const*>(substr);

  *nb_retval = str_view->rfind(*substr_view);
  return 0;
}

extern "C" __device__ int eq(bool* nb_retval, void const* str, void const* rhs)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);
  auto rhs_view = reinterpret_cast<cudf::string_view const*>(rhs);

  *nb_retval = (*str_view == *rhs_view);
  return 0;
}

extern "C" __device__ int ne(bool* nb_retval, void const* str, void const* rhs)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);
  auto rhs_view = reinterpret_cast<cudf::string_view const*>(rhs);

  *nb_retval = (*str_view != *rhs_view);
  return 0;
}

extern "C" __device__ int ge(bool* nb_retval, void const* str, void const* rhs)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);
  auto rhs_view = reinterpret_cast<cudf::string_view const*>(rhs);

  *nb_retval = (*str_view >= *rhs_view);
  return 0;
}

extern "C" __device__ int le(bool* nb_retval, void const* str, void const* rhs)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);
  auto rhs_view = reinterpret_cast<cudf::string_view const*>(rhs);

  *nb_retval = (*str_view <= *rhs_view);
  return 0;
}

extern "C" __device__ int gt(bool* nb_retval, void const* str, void const* rhs)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);
  auto rhs_view = reinterpret_cast<cudf::string_view const*>(rhs);

  *nb_retval = (*str_view > *rhs_view);
  return 0;
}

extern "C" __device__ int lt(bool* nb_retval, void const* str, void const* rhs)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);
  auto rhs_view = reinterpret_cast<cudf::string_view const*>(rhs);

  *nb_retval = (*str_view < *rhs_view);
  return 0;
}

extern "C" __device__ int pyislower(bool* nb_retval, void const* str, std::uintptr_t chars_table)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);

  *nb_retval = is_lower(
    reinterpret_cast<cudf::strings::detail::character_flags_table_type*>(chars_table), *str_view);
  return 0;
}

extern "C" __device__ int pyisupper(bool* nb_retval, void const* str, std::uintptr_t chars_table)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);

  *nb_retval = is_upper(
    reinterpret_cast<cudf::strings::detail::character_flags_table_type*>(chars_table), *str_view);
  return 0;
}

extern "C" __device__ int pyisspace(bool* nb_retval, void const* str, std::uintptr_t chars_table)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);

  *nb_retval = is_space(
    reinterpret_cast<cudf::strings::detail::character_flags_table_type*>(chars_table), *str_view);
  return 0;
}

extern "C" __device__ int pyisdecimal(bool* nb_retval, void const* str, std::uintptr_t chars_table)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);

  *nb_retval = is_decimal(
    reinterpret_cast<cudf::strings::detail::character_flags_table_type*>(chars_table), *str_view);
  return 0;
}

extern "C" __device__ int pyisnumeric(bool* nb_retval, void const* str, std::uintptr_t chars_table)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);

  *nb_retval = is_numeric(
    reinterpret_cast<cudf::strings::detail::character_flags_table_type*>(chars_table), *str_view);
  return 0;
}

extern "C" __device__ int pyisdigit(bool* nb_retval, void const* str, std::uintptr_t chars_table)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);

  *nb_retval = is_digit(
    reinterpret_cast<cudf::strings::detail::character_flags_table_type*>(chars_table), *str_view);
  return 0;
}

extern "C" __device__ int pyisalnum(bool* nb_retval, void const* str, std::uintptr_t chars_table)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);

  *nb_retval = is_alpha_numeric(
    reinterpret_cast<cudf::strings::detail::character_flags_table_type*>(chars_table), *str_view);
  return 0;
}

extern "C" __device__ int pyisalpha(bool* nb_retval, void const* str, std::uintptr_t chars_table)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);

  *nb_retval = is_alpha(
    reinterpret_cast<cudf::strings::detail::character_flags_table_type*>(chars_table), *str_view);
  return 0;
}

extern "C" __device__ int pyistitle(bool* nb_retval, void const* str, std::uintptr_t chars_table)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);

  *nb_retval = is_title(
    reinterpret_cast<cudf::strings::detail::character_flags_table_type*>(chars_table), *str_view);
  return 0;
}

extern "C" __device__ int pycount(int* nb_retval, void const* str, void const* substr)
{
  auto str_view    = reinterpret_cast<cudf::string_view const*>(str);
  auto substr_view = reinterpret_cast<cudf::string_view const*>(substr);

  *nb_retval = count(*str_view, *substr_view);
  return 0;
}

extern "C" __device__ int udf_string_from_string_view(int* nb_retbal,
                                                      void const* str,
                                                      void* udf_str)
{
  auto str_view_ptr = reinterpret_cast<cudf::string_view const*>(str);
  auto udf_str_ptr  = new (udf_str) udf_string;
  *udf_str_ptr      = udf_string(*str_view_ptr);

  return 0;
}

extern "C" __device__ int strip(int* nb_retval,
                                void* udf_str,
                                void* const* to_strip,
                                void* const* strip_str)
{
  auto to_strip_ptr  = reinterpret_cast<cudf::string_view const*>(to_strip);
  auto strip_str_ptr = reinterpret_cast<cudf::string_view const*>(strip_str);
  auto udf_str_ptr   = new (udf_str) udf_string;

  *udf_str_ptr = strip(*to_strip_ptr, *strip_str_ptr);

  return 0;
}

extern "C" __device__ int lstrip(int* nb_retval,
                                 void* udf_str,
                                 void* const* to_strip,
                                 void* const* strip_str)
{
  auto to_strip_ptr  = reinterpret_cast<cudf::string_view const*>(to_strip);
  auto strip_str_ptr = reinterpret_cast<cudf::string_view const*>(strip_str);
  auto udf_str_ptr   = new (udf_str) udf_string;

  *udf_str_ptr = strip(*to_strip_ptr, *strip_str_ptr, cudf::strings::side_type::LEFT);

  return 0;
}

extern "C" __device__ int rstrip(int* nb_retval,
                                 void* udf_str,
                                 void* const* to_strip,
                                 void* const* strip_str)
{
  auto to_strip_ptr  = reinterpret_cast<cudf::string_view const*>(to_strip);
  auto strip_str_ptr = reinterpret_cast<cudf::string_view const*>(strip_str);
  auto udf_str_ptr   = new (udf_str) udf_string;

  *udf_str_ptr = strip(*to_strip_ptr, *strip_str_ptr, cudf::strings::side_type::RIGHT);

  return 0;
}
extern "C" __device__ int upper(int* nb_retval,
                                void* udf_str,
                                void const* st,
                                std::uintptr_t flags_table,
                                std::uintptr_t cases_table,
                                std::uintptr_t special_table)
{
  auto udf_str_ptr = new (udf_str) udf_string;
  auto st_ptr      = reinterpret_cast<cudf::string_view const*>(st);

  auto flags_table_ptr =
    reinterpret_cast<cudf::strings::detail::character_flags_table_type*>(flags_table);
  auto cases_table_ptr =
    reinterpret_cast<cudf::strings::detail::character_cases_table_type*>(cases_table);
  auto special_table_ptr =
    reinterpret_cast<cudf::strings::detail::special_case_mapping*>(special_table);

  cudf::strings::udf::chars_tables tables{flags_table_ptr, cases_table_ptr, special_table_ptr};

  *udf_str_ptr = to_upper(tables, *st_ptr);

  return 0;
}

extern "C" __device__ int lower(int* nb_retval,
                                void* udf_str,
                                void const* st,
                                std::uintptr_t flags_table,
                                std::uintptr_t cases_table,
                                std::uintptr_t special_table)
{
  auto udf_str_ptr = new (udf_str) udf_string;
  auto st_ptr      = reinterpret_cast<cudf::string_view const*>(st);

  auto flags_table_ptr =
    reinterpret_cast<cudf::strings::detail::character_flags_table_type*>(flags_table);
  auto cases_table_ptr =
    reinterpret_cast<cudf::strings::detail::character_cases_table_type*>(cases_table);
  auto special_table_ptr =
    reinterpret_cast<cudf::strings::detail::special_case_mapping*>(special_table);

  cudf::strings::udf::chars_tables tables{flags_table_ptr, cases_table_ptr, special_table_ptr};
  *udf_str_ptr = to_lower(tables, *st_ptr);
  return 0;
}

extern "C" __device__ int concat(int* nb_retval, void* udf_str, void* const* lhs, void* const* rhs)
{
  auto lhs_ptr = reinterpret_cast<cudf::string_view const*>(lhs);
  auto rhs_ptr = reinterpret_cast<cudf::string_view const*>(rhs);

  auto udf_str_ptr = new (udf_str) udf_string;

  udf_string result;
  result.append(*lhs_ptr).append(*rhs_ptr);
  *udf_str_ptr = result;
  return 0;
}

extern "C" __device__ int replace(
  int* nb_retval, void* udf_str, void* const src, void* const to_replace, void* const replacement)
{
  auto src_ptr         = reinterpret_cast<cudf::string_view const*>(src);
  auto to_replace_ptr  = reinterpret_cast<cudf::string_view const*>(to_replace);
  auto replacement_ptr = reinterpret_cast<cudf::string_view const*>(replacement);

  auto udf_str_ptr = new (udf_str) udf_string;
  *udf_str_ptr     = replace(*src_ptr, *to_replace_ptr, *replacement_ptr);

  return 0;
}

// Groupby Shim Functions
template <typename T>
__device__ bool are_all_nans(cooperative_groups::thread_block const& block,
                             T const* data,
                             int64_t size)
{
  // TODO: to be refactored with CG vote functions once
  // block size is known at build time
  __shared__ int64_t count;

  if (block.thread_rank() == 0) { count = 0; }
  block.sync();

  for (int64_t idx = block.thread_rank(); idx < size; idx += block.size()) {
    if (not std::isnan(data[idx])) {
      cuda::atomic_ref<int64_t, cuda::thread_scope_block> ref{count};
      ref.fetch_add(1, cuda::std::memory_order_relaxed);
      break;
    }
  }

  block.sync();
  return count == 0;
}

template <typename T>
__device__ void device_sum(cooperative_groups::thread_block const& block,
                           T const* data,
                           int64_t size,
                           T* sum)
{
  T local_sum = 0;

  for (int64_t idx = block.thread_rank(); idx < size; idx += block.size()) {
    local_sum += data[idx];
  }

  cuda::atomic_ref<T, cuda::thread_scope_block> ref{*sum};
  ref.fetch_add(local_sum, cuda::std::memory_order_relaxed);

  block.sync();
}

template <typename T>
__device__ T BlockSum(T const* data, int64_t size)
{
  auto block = cooperative_groups::this_thread_block();

  if constexpr (std::is_floating_point_v<T>) {
    if (are_all_nans(block, data, size)) { return 0; }
  }

  __shared__ T block_sum;
  if (block.thread_rank() == 0) { block_sum = 0; }
  block.sync();

  device_sum<T>(block, data, size, &block_sum);
  return block_sum;
}

template <typename T>
__device__ double BlockMean(T const* data, int64_t size)
{
  auto block = cooperative_groups::this_thread_block();

  __shared__ T block_sum;
  if (block.thread_rank() == 0) { block_sum = 0; }
  block.sync();

  device_sum<T>(block, data, size, &block_sum);
  return static_cast<double>(block_sum) / static_cast<double>(size);
}

template <typename T>
__device__ double BlockVar(T const* data, int64_t size)
{
  auto block = cooperative_groups::this_thread_block();

  __shared__ double block_var;
  __shared__ T block_sum;
  if (block.thread_rank() == 0) {
    block_var = 0;
    block_sum = 0;
  }
  block.sync();

  T local_sum      = 0;
  double local_var = 0;

  device_sum<T>(block, data, size, &block_sum);

  auto const mean = static_cast<double>(block_sum) / static_cast<double>(size);

  for (int64_t idx = block.thread_rank(); idx < size; idx += block.size()) {
    auto const delta = static_cast<double>(data[idx]) - mean;
    local_var += delta * delta;
  }

  cuda::atomic_ref<double, cuda::thread_scope_block> ref{block_var};
  ref.fetch_add(local_var, cuda::std::memory_order_relaxed);
  block.sync();

  if (block.thread_rank() == 0) { block_var = block_var / static_cast<double>(size - 1); }
  block.sync();
  return block_var;
}

template <typename T>
__device__ double BlockStd(T const* data, int64_t size)
{
  auto const var = BlockVar(data, size);
  return sqrt(var);
}

template <typename T>
__device__ T BlockMax(T const* data, int64_t size)
{
  auto block = cooperative_groups::this_thread_block();

  if constexpr (std::is_floating_point_v<T>) {
    if (are_all_nans(block, data, size)) { return std::numeric_limits<T>::quiet_NaN(); }
  }

  auto local_max = cudf::DeviceMax::identity<T>();
  __shared__ T block_max;
  if (block.thread_rank() == 0) { block_max = local_max; }
  block.sync();

  for (int64_t idx = block.thread_rank(); idx < size; idx += block.size()) {
    local_max = max(local_max, data[idx]);
  }

  cuda::atomic_ref<T, cuda::thread_scope_block> ref{block_max};
  ref.fetch_max(local_max, cuda::std::memory_order_relaxed);

  block.sync();

  return block_max;
}

template <typename T>
__device__ T BlockMin(T const* data, int64_t size)
{
  auto block = cooperative_groups::this_thread_block();

  if constexpr (std::is_floating_point_v<T>) {
    if (are_all_nans(block, data, size)) { return std::numeric_limits<T>::quiet_NaN(); }
  }

  auto local_min = cudf::DeviceMin::identity<T>();

  __shared__ T block_min;
  if (block.thread_rank() == 0) { block_min = local_min; }
  block.sync();

  for (int64_t idx = block.thread_rank(); idx < size; idx += block.size()) {
    local_min = min(local_min, data[idx]);
  }

  cuda::atomic_ref<T, cuda::thread_scope_block> ref{block_min};
  ref.fetch_min(local_min, cuda::std::memory_order_relaxed);

  block.sync();

  return block_min;
}

template <typename T>
__device__ int64_t BlockIdxMax(T const* data, int64_t* index, int64_t size)
{
  auto block = cooperative_groups::this_thread_block();

  __shared__ T block_max;
  __shared__ int64_t block_idx_max;
  __shared__ bool found_max;

  auto local_max     = cudf::DeviceMax::identity<T>();
  auto local_idx_max = cudf::DeviceMin::identity<int64_t>();

  if (block.thread_rank() == 0) {
    block_max     = local_max;
    block_idx_max = local_idx_max;
    found_max     = false;
  }
  block.sync();

  for (int64_t idx = block.thread_rank(); idx < size; idx += block.size()) {
    auto const current_data = data[idx];
    if (current_data > local_max) {
      local_max     = current_data;
      local_idx_max = index[idx];
      found_max     = true;
    }
  }

  cuda::atomic_ref<T, cuda::thread_scope_block> ref{block_max};
  ref.fetch_max(local_max, cuda::std::memory_order_relaxed);
  block.sync();

  if (found_max) {
    if (local_max == block_max) {
      cuda::atomic_ref<int64_t, cuda::thread_scope_block> ref_idx{block_idx_max};
      ref_idx.fetch_min(local_idx_max, cuda::std::memory_order_relaxed);
    }
  } else {
    if (block.thread_rank() == 0) { block_idx_max = index[0]; }
  }
  block.sync();

  return block_idx_max;
}

template <typename T>
__device__ int64_t BlockIdxMin(T const* data, int64_t* index, int64_t size)
{
  auto block = cooperative_groups::this_thread_block();

  __shared__ T block_min;
  __shared__ int64_t block_idx_min;
  __shared__ bool found_min;

  auto local_min     = cudf::DeviceMin::identity<T>();
  auto local_idx_min = cudf::DeviceMin::identity<int64_t>();

  if (block.thread_rank() == 0) {
    block_min     = local_min;
    block_idx_min = local_idx_min;
    found_min     = false;
  }
  block.sync();

  for (int64_t idx = block.thread_rank(); idx < size; idx += block.size()) {
    auto const current_data = data[idx];
    if (current_data < local_min) {
      local_min     = current_data;
      local_idx_min = index[idx];
      found_min     = true;
    }
  }

  cuda::atomic_ref<T, cuda::thread_scope_block> ref{block_min};
  ref.fetch_min(local_min, cuda::std::memory_order_relaxed);
  block.sync();

  if (found_min) {
    if (local_min == block_min) {
      cuda::atomic_ref<int64_t, cuda::thread_scope_block> ref_idx{block_idx_min};
      ref_idx.fetch_min(local_idx_min, cuda::std::memory_order_relaxed);
    }
  } else {
    if (block.thread_rank() == 0) { block_idx_min = index[0]; }
  }
  block.sync();

  return block_idx_min;
}

extern "C" {
#define make_definition(name, cname, type, return_type)                                          \
  __device__ int name##_##cname(return_type* numba_return_value, type* const data, int64_t size) \
  {                                                                                              \
    return_type const res = name<type>(data, size);                                              \
    *numba_return_value   = res;                                                                 \
    __syncthreads();                                                                             \
    return 0;                                                                                    \
  }

make_definition(BlockSum, int64, int64_t, int64_t);
make_definition(BlockSum, float64, double, double);
make_definition(BlockMean, int64, int64_t, double);
make_definition(BlockMean, float64, double, double);
make_definition(BlockStd, int64, int64_t, double);
make_definition(BlockStd, float64, double, double);
make_definition(BlockVar, int64, int64_t, double);
make_definition(BlockVar, float64, double, double);
make_definition(BlockMin, int64, int64_t, int64_t);
make_definition(BlockMin, float64, double, double);
make_definition(BlockMax, int64, int64_t, int64_t);
make_definition(BlockMax, float64, double, double);
#undef make_definition
}

extern "C" {
#define make_definition_idx(name, cname, type)                                   \
  __device__ int name##_##cname(                                                 \
    int64_t* numba_return_value, type* const data, int64_t* index, int64_t size) \
  {                                                                              \
    auto const res      = name<type>(data, index, size);                         \
    *numba_return_value = res;                                                   \
    __syncthreads();                                                             \
    return 0;                                                                    \
  }

make_definition_idx(BlockIdxMin, int64, int64_t);
make_definition_idx(BlockIdxMin, float64, double);
make_definition_idx(BlockIdxMax, int64, int64_t);
make_definition_idx(BlockIdxMax, float64, double);
#undef make_definition_idx
}

// Reference Counting
// nrt.h port, required for nrt.cpp
typedef void (*NRT_dtor_function)(void* ptr, size_t size, void* info);
typedef void (*NRT_dealloc_func)(void* ptr, void* dealloc_info);

typedef void* (*NRT_malloc_func)(size_t size);
typedef void* (*NRT_realloc_func)(void* ptr, size_t new_size);
typedef void (*NRT_free_func)(void* ptr);

// nrt.cpp port
extern "C" {
struct CUDAMemInfo {
  cuda::atomic<size_t> refct;
  NRT_dtor_function dtor;
  void* data;
  size_t size;
};
}

typedef struct CUDAMemInfo CUDANRT_MemInfo;

// vendored directly from nrt.cpp
struct CUDANRT_MemSys {
  /* Shutdown flag */
  int shutting;
  /* Stats */
  struct {
    bool enabled;
    cuda::atomic<size_t> alloc;
    cuda::atomic<size_t> free;
    cuda::atomic<size_t> mi_alloc;
    cuda::atomic<size_t> mi_free;
  } stats;
  /* System allocation functions */
  struct {
    NRT_malloc_func malloc;
    NRT_realloc_func realloc;
    NRT_free_func free;
  } allocator;
};

/* The Memory System object */
__device__ static CUDANRT_MemSys TheMSys;

static void nrt_fatal_error(const char* msg)
{
  fprintf(stderr, "Fatal Numba error: %s\n", msg);
  fflush(stderr); /* it helps in Windows debug build */
}

extern "C" __device__ void CUDANRT_MemSys_init(void)
{
  TheMSys.shutting = 0;
  // Stats are off by default, call NRT_MemSys_enable_stats to enable
  TheMSys.stats.enabled  = false;
  TheMSys.stats.alloc    = 0;
  TheMSys.stats.free     = 0;
  TheMSys.stats.mi_alloc = 0;
  TheMSys.stats.mi_free  = 0;
  /* Bind to CUDA allocator */
  TheMSys.allocator.malloc = malloc;
  //  TheMSys.allocator.realloc = realloc;
  TheMSys.allocator.free = free;
}

extern "C" void CUDANRT_MemSys_shutdown(void) { TheMSys.shutting = 1; }

extern "C" void CUDANRT_MemSys_enable_stats(void) { TheMSys.stats.enabled = true; }

extern "C" void CUDANRT_MemSys_disable_stats(void) { TheMSys.stats.enabled = false; }

extern "C" size_t CUDANRT_MemSys_stats_enabled(void) { return (size_t)TheMSys.stats.enabled; }

extern "C" void NRT_MemSys_set_allocator(NRT_malloc_func malloc_func,
                                         NRT_realloc_func realloc_func,
                                         NRT_free_func free_func)
{
  bool stats_cond = false;
  if ((malloc_func != TheMSys.allocator.malloc ||
       // realloc_func != TheMSys.allocator.realloc ||
       free_func != TheMSys.allocator.free) &&
      stats_cond) {
    nrt_fatal_error("cannot change allocator while blocks are allocated");
  }
  TheMSys.allocator.malloc = malloc_func;
  // TheMSys.allocator.realloc = realloc_func;
  TheMSys.allocator.free = free_func;
}

/* This value is used as a marker for "stats are disabled", it's ASCII "AAAA" */
static size_t _DISABLED_STATS_VALUE = 0x41414141;

extern "C" size_t NRT_MemSys_get_stats_alloc()
{
  if (TheMSys.stats.enabled) {
    return TheMSys.stats.alloc.load();
  } else {
    return _DISABLED_STATS_VALUE;
  }
}

extern "C" size_t NRT_MemSys_get_stats_free()
{
  if (TheMSys.stats.enabled) {
    return TheMSys.stats.free.load();
  } else {
    return _DISABLED_STATS_VALUE;
  }
}

extern "C" size_t NRT_MemSys_get_stats_mi_alloc()
{
  if (TheMSys.stats.enabled) {
    return TheMSys.stats.mi_alloc.load();
  } else {
    return _DISABLED_STATS_VALUE;
  }
}

extern "C" size_t NRT_MemSys_get_stats_mi_free()
{
  if (TheMSys.stats.enabled) {
    return TheMSys.stats.mi_free.load();
  } else {
    return _DISABLED_STATS_VALUE;
  }
}

/*
 * The MemInfo structure.
 */

extern "C" __device__ void CUDANRT_MemInfo_init(CUDAMemInfo* mi,
                                                void* data,
                                                size_t size,
                                                NRT_dtor_function dtor)
{
  mi->refct = 1; /* starts with 1 refct */
  mi->dtor  = dtor;
  mi->data  = data;
  mi->size  = size;
  /* Update stats */
  if (TheMSys.stats.enabled) { TheMSys.stats.mi_alloc++; }
}

extern "C" __device__ void* CUDANRT_Allocate_External(size_t size)
{
  void* ptr = NULL;
  ptr       = TheMSys.allocator.malloc(size);
  if (TheMSys.stats.enabled) { TheMSys.stats.alloc++; }
  return ptr;
}

extern "C" __device__ void* CUDANRT_Allocate(size_t size)
{
  return CUDANRT_Allocate_External(size);
}

__device__ CUDANRT_MemInfo* CUDANRT_MemInfo_new(void* data,
                                                size_t size,
                                                NRT_dtor_function dtor,
                                                void* dtor_info)
{
  CUDANRT_MemInfo* mi = (CUDANRT_MemInfo*)CUDANRT_Allocate(sizeof(CUDANRT_MemInfo));
  if (mi != NULL) { CUDANRT_MemInfo_init(mi, data, size, dtor); }
  return mi;
}

__device__ size_t CUDANRT_MemInfo_refcount(CUDANRT_MemInfo* mi)
{
  /* Should never returns 0 for a valid MemInfo */
  if (mi && mi->data)
    return mi->refct;
  else {
    return (size_t)-1;
  }
}

__device__ static void* nrt_allocate_meminfo_and_data(size_t size, CUDANRT_MemInfo** mi_out)
{
  CUDANRT_MemInfo* mi = NULL;
  char* base          = (char*)CUDANRT_Allocate_External(sizeof(CUDANRT_MemInfo) + size);
  if (base == NULL) {
    *mi_out = NULL; /* set meminfo to NULL as allocation failed */
    return NULL;    /* return early as allocation failed */
  }
  mi      = (CUDANRT_MemInfo*)base;
  *mi_out = mi;
  return (void*)((char*)base + sizeof(CUDANRT_MemInfo));
}

__device__ CUDANRT_MemInfo* CUDANRT_MemInfo_alloc(size_t size)
{
  CUDANRT_MemInfo* mi = NULL;
  void* data          = nrt_allocate_meminfo_and_data(size, &mi);
  if (data == NULL) { return NULL; /* return early as allocation failed */ }
  CUDANRT_MemInfo_init(mi, data, size, NULL);
  return mi;
}

__device__ static void nrt_internal_custom_dtor(void* ptr, size_t size, void* info)
{
  NRT_dtor_function dtor = (NRT_dtor_function)info;
  if (dtor) { dtor(ptr, size, NULL); }
}

__device__ CUDANRT_MemInfo* CUDANRT_MemInfo_alloc_dtor(size_t size, NRT_dtor_function dtor)
{
  CUDANRT_MemInfo* mi = NULL;
  void* data          = (void*)nrt_allocate_meminfo_and_data(size, &mi);
  if (data == NULL) { return NULL; /* return early as allocation failed */ }
  CUDANRT_MemInfo_init(mi, data, size, nrt_internal_custom_dtor);
  return mi;
}

__device__ static void* nrt_allocate_meminfo_and_data_align(size_t size,
                                                            unsigned align,
                                                            CUDANRT_MemInfo** mi)
{
  size_t offset = 0, intptr = 0, remainder = 0;
  char* base = (char*)nrt_allocate_meminfo_and_data(size + 2 * align, mi);
  if (base == NULL) { return NULL; /* return early as allocation failed */ }
  intptr = (size_t)base;
  /*
   * See if the allocation is aligned already...
   * Check if align is a power of 2, if so the modulo can be avoided.
   */
  if ((align & (align - 1)) == 0) {
    remainder = intptr & (align - 1);
  } else {
    remainder = intptr % align;
  }
  if (remainder == 0) { /* Yes */
    offset = 0;
  } else { /* No, move forward `offset` bytes */
    offset = align - remainder;
  }
  return (void*)((char*)base + offset);
}

extern "C" __device__ CUDANRT_MemInfo* NRT_MemInfo_alloc_aligned(size_t size, unsigned align)
{
  CUDANRT_MemInfo* mi = NULL;
  void* data          = nrt_allocate_meminfo_and_data_align(size, align, &mi);
  if (data == NULL) { return NULL; /* return early as allocation failed */ }
  CUDANRT_MemInfo_init(mi, data, size, NULL);
  return mi;
}

extern "C" __device__ void NRT_Free(void* ptr) { TheMSys.allocator.free(ptr); }

extern "C" __device__ void CUDANRT_dealloc(CUDANRT_MemInfo* mi) { NRT_Free(mi); }

typedef void NRT_managed_dtor(void* data);

__device__ static void nrt_manage_memory_dtor(void* data, size_t size, void* info)
{
  NRT_managed_dtor* dtor = (NRT_managed_dtor*)info;
  dtor(data);
}

__device__ static CUDANRT_MemInfo* nrt_manage_memory(void* data, NRT_managed_dtor dtor)
{
  return (CUDANRT_MemInfo*)(CUDANRT_MemInfo_new(data, 0, nrt_manage_memory_dtor, (void*)dtor));
}

typedef struct {
  /* Methods to create MemInfos.
  MemInfos are like smart pointers for objects that are managed by the Numba.
  */

  /* Allocate memory
  *nbytes* is the number of bytes to be allocated
  Returning a new reference.
  */
  CUDANRT_MemInfo* (*allocate)(size_t nbytes);

  /* Convert externally allocated memory into a MemInfo.
   *data* is the memory pointer
   *dtor* is the deallocator of the memory
   */
  CUDANRT_MemInfo* (*manage_memory)(void* data, NRT_managed_dtor dtor);

  /* Acquire a reference */
  void (*acquire)(CUDANRT_MemInfo* mi);

  /* Release a reference */
  void (*release)(CUDANRT_MemInfo* mi);

  /* Get MemInfo data pointer */
  void* (*get_data)(CUDANRT_MemInfo* mi);

} NRT_api_functions;

extern "C" __device__ void NRT_MemInfo_destroy(CUDANRT_MemInfo* mi) { CUDANRT_dealloc(mi); }

extern "C" __device__ void NRT_MemInfo_call_dtor(CUDANRT_MemInfo* mi)
{
  if (mi->dtor && !TheMSys.shutting) /* We have a destructor and the system is not shutting down */
    mi->dtor(mi->data, mi->size, NULL);
  /* Clear and release MemInfo */
  NRT_MemInfo_destroy(mi);
}

extern "C" __device__ void NRT_MemInfo_acquire(CUDANRT_MemInfo* mi)
{
  assert(mi->refct > 0 && "RefCt cannot be zero");
  mi->refct++;
}

extern "C" __device__ void NRT_MemInfo_release(CUDANRT_MemInfo* mi)
{
  assert(mi->refct > 0 && "RefCt cannot be 0");
  /* RefCt drop to zero */
  if ((--(mi->refct)) == 0) { NRT_MemInfo_call_dtor(mi); }
}

extern "C" __device__ void* NRT_MemInfo_data(CUDANRT_MemInfo* mi) { return mi->data; }

static const NRT_api_functions nrt_functions_table = {CUDANRT_MemInfo_alloc,
                                                      nrt_manage_memory,
                                                      NRT_MemInfo_acquire,
                                                      NRT_MemInfo_release,
                                                      NRT_MemInfo_data};

extern "C" const NRT_api_functions* CUDANRT_get_api(void) { return &nrt_functions_table; }
